#include "cctk.h"
#include "cctk_Parameters.h"
#include "cctk_Arguments.h"

#include "CaCUDALib.h"
#include "CaCUDAUtil.h"

#ifdef __HIPCC__
#  include <hip/hip_runtime.h>
#  include <hip/hip_runtime.h>
#endif

/* Check that the problem size will give the best performance
 *
 * This will need the device properties and problem size to give
 * the best guess.
 * */

void CaCUDALib_ParamCheck (CCTK_ARGUMENTS)
{
	DECLARE_CCTK_ARGUMENTS;
	DECLARE_CCTK_PARAMETERS;

/*get attributes of the registered Kernel function */
	  struct hipFuncAttributes funcAttrib;
	  CUDA_SAFE_CALL(hipFuncGetAttributes(&funcAttrib, reinterpret_cast<const void*>(globalfunc)));

}
